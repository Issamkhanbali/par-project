#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void rnn_kernel(float *data, float *weights_input, float *weights_hidden, float *bias, int input_size, int hidden_size, int data_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < data_size) {
        int timestep = idx / input_size;
        if (timestep > 0) {
            for (int i = 0; i < hidden_size; ++i) {
                float output = bias[i];
                for (int j = 0; j < input_size; ++j) {
                    output += data[idx + j] * weights_input[j * hidden_size + i];
                }
                for (int j = 0; j < hidden_size; ++j) {
                    output += data[(timestep - 1) * hidden_size + j] * weights_hidden[j * hidden_size + i];
                }
                data[timestep * hidden_size + i] = tanh(output);
            }
        }
    }
}

int main() {
    int input_size = 3;
    int hidden_size = 4;
    int timesteps = 10;
    int data_size = timesteps * input_size;

    float *h_data = (float *)malloc(data_size * sizeof(float));
    float *h_weights_input = (float *)malloc(input_size * hidden_size * sizeof(float));
    float *h_weights_hidden = (float *)malloc(hidden_size * hidden_size * sizeof(float));
    float *h_bias = (float *)malloc(hidden_size * sizeof(float));

   
    for (int i = 0; i < data_size; ++i) {
        h_data[i] = (float)rand() / RAND_MAX;
    }
    for (int i = 0; i < input_size * hidden_size; ++i) {
        h_weights_input[i] = (float)rand() / RAND_MAX;
    }
    for (int i = 0; i < hidden_size * hidden_size; ++i) {
        h_weights_hidden[i] = (float)rand() / RAND_MAX;
    }
    for (int i = 0; i < hidden_size; ++i) {
        h_bias[i] = (float)rand() / RAND_MAX;
    }

    float *d_data, *d_weights_input, *d_weights_hidden, *d_bias;
    hipMalloc(&d_data, data_size * sizeof(float));
    hipMalloc(&d_weights_input, input_size * hidden_size * sizeof(float));
    hipMalloc(&d_weights_hidden, hidden_size * hidden_size * sizeof(float));
    hipMalloc(&d_bias, hidden_size * sizeof(float));

    hipMemcpy(d_data, h_data, data_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weights_input, h_weights_input, input_size * hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weights_hidden, h_weights_hidden, hidden_size * hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, h_bias, hidden_size * sizeof(float), hipMemcpyHostToDevice);

    
    int blockSize = 1;  
    int numBlocks = 1;
    rnn_kernel<<<numBlocks, blockSize>>>(d_data, d_weights_input, d_weights_hidden, d_bias, input_size, hidden_size, data_size);

    
    blockSize = 2;
    rnn_kernel<<<numBlocks, blockSize>>>(d_data, d_weights_input, d_weights_hidden, d_bias, input_size, hidden_size, data_size);

   
    blockSize = 4;
    rnn_kernel<<<numBlocks, blockSize>>>(d_data, d_weights_input, d_weights_hidden, d_bias, input_size, hidden_size, data_size);

    hipMemcpy(h_data, d_data, data_size * sizeof(float), hipMemcpyDeviceToHost);

    free(h_data);
    free(h_weights_input);
    free(h_weights_hidden);
    free(h_bias);
    hipFree(d_data);
    hipFree(d_weights_input);
    hipFree(d_weights_hidden);
    hipFree(d_bias);

    return 0;
}
